#include "hip/hip_runtime.h"
#include "op_cuda.h"
#include <cmath>

// A helper for launching kernels
static inline int getGridSize(int n, int block_size=256) {
    return (n + block_size - 1) / block_size;
}

// -------------------- Add --------------------
__global__ void add_forward_kernel(const float* a, const float* b, float* out, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        out[idx] = a[idx] + b[idx];
    }
}

void add_forward_cuda(const float* a, const float* b, float* out, int size) {
    int grid = getGridSize(size);
    add_forward_kernel<<<grid, 256>>>(a, b, out, size);
    hipDeviceSynchronize();
}

__global__ void add_backward_kernel(const float* grad_out, float* grad_a, float* grad_b, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        grad_a[idx] += grad_out[idx];
        grad_b[idx] += grad_out[idx];
    }
}

void add_backward_cuda(const float* grad_out, float* grad_a, float* grad_b, int size) {
    int grid = getGridSize(size);
    add_backward_kernel<<<grid, 256>>>(grad_out, grad_a, grad_b, size);
    hipDeviceSynchronize();
}

// -------------------- Subtract --------------------
__global__ void sub_forward_kernel(const float* a, const float* b, float* out, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        out[idx] = a[idx] - b[idx];
    }
}

void sub_forward_cuda(const float* a, const float* b, float* out, int size) {
    int grid = getGridSize(size);
    sub_forward_kernel<<<grid, 256>>>(a, b, out, size);
    hipDeviceSynchronize();
}

__global__ void sub_backward_kernel(const float* grad_out, float* grad_a, float* grad_b, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        grad_a[idx] += grad_out[idx];
        grad_b[idx] -= grad_out[idx];
    }
}

void sub_backward_cuda(const float* grad_out, float* grad_a, float* grad_b, int size) {
    int grid = getGridSize(size);
    sub_backward_kernel<<<grid, 256>>>(grad_out, grad_a, grad_b, size);
    hipDeviceSynchronize();
}

// -------------------- Multiply --------------------
__global__ void mul_forward_kernel(const float* a, const float* b, float* out, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        out[idx] = a[idx] * b[idx];
    }
}

void mul_forward_cuda(const float* a, const float* b, float* out, int size) {
    int grid = getGridSize(size);
    mul_forward_kernel<<<grid, 256>>>(a, b, out, size);
    hipDeviceSynchronize();
}

__global__ void mul_backward_kernel(const float* grad_out, const float* a, const float* b, float* grad_a, float* grad_b, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        grad_a[idx] += grad_out[idx] * b[idx];
        grad_b[idx] += grad_out[idx] * a[idx];
    }
}

void mul_backward_cuda(const float* grad_out, const float* a, const float* b, float* grad_a, float* grad_b, int size) {
    int grid = getGridSize(size);
    mul_backward_kernel<<<grid, 256>>>(grad_out, a, b, grad_a, grad_b, size);
    hipDeviceSynchronize();
}

// -------------------- Divide --------------------
__global__ void div_forward_kernel(const float* a, const float* b, float* out, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float denom = b[idx];
        // Assume no zero division as checked in CPU code
        out[idx] = a[idx] / denom;
    }
}

void div_forward_cuda(const float* a, const float* b, float* out, int size) {
    int grid = getGridSize(size);
    div_forward_kernel<<<grid, 256>>>(a, b, out, size);
    hipDeviceSynchronize();
}

__global__ void div_backward_kernel(const float* grad_out, const float* a, const float* b, float* grad_a, float* grad_b, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float A = a[idx];
        float B = b[idx];
        grad_a[idx] += grad_out[idx] / B;
        grad_b[idx] -= (A * grad_out[idx]) / (B * B);
    }
}

void div_backward_cuda(const float* grad_out, const float* a, const float* b, float* grad_a, float* grad_b, int size) {
    int grid = getGridSize(size);
    div_backward_kernel<<<grid, 256>>>(grad_out, a, b, grad_a, grad_b, size);
    hipDeviceSynchronize();
}

// -------------------- Exp --------------------
__global__ void exp_forward_kernel(const float* a, float* out, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        out[idx] = expf(a[idx]);
    }
}

void exp_forward_cuda(const float* a, float* out, int size) {
    int grid = getGridSize(size);
    exp_forward_kernel<<<grid, 256>>>(a, out, size);
    hipDeviceSynchronize();
}

__global__ void exp_backward_kernel(const float* grad_out, const float* a, float* grad_a, int size) {
    // Need to recompute exp(a)
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float val = expf(a[idx]);
        grad_a[idx] += grad_out[idx] * val;
    }
}

void exp_backward_cuda(const float* grad_out, const float* a, float* grad_a, int size) {
    int grid = getGridSize(size);
    exp_backward_kernel<<<grid, 256>>>(grad_out, a, grad_a, size);
    hipDeviceSynchronize();
}

// -------------------- Tanh --------------------
// We'll assume we have 'out' as the forward output passed to backward for simplicity.
// If not, you'd need to recompute out = tanh(a[idx]) in backward.
__global__ void tanh_forward_kernel(const float* a, float* out, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        out[idx] = tanhf(a[idx]);
    }
}

void tanh_forward_cuda(const float* a, float* out, int size) {
    int grid = getGridSize(size);
    tanh_forward_kernel<<<grid, 256>>>(a, out, size);
    hipDeviceSynchronize();
}

__global__ void tanh_backward_kernel(const float* grad_out, const float* out, float* grad_a, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float t = out[idx];
        grad_a[idx] += grad_out[idx] * (1.0f - t * t);
    }
}

void tanh_backward_cuda(const float* grad_out, const float* out, float* grad_a, int size) {
    int grid = getGridSize(size);
    tanh_backward_kernel<<<grid, 256>>>(grad_out, out, grad_a, size);
    hipDeviceSynchronize();
}

// -------------------- Relu --------------------
__global__ void relu_forward_kernel(const float* a, float* out, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        out[idx] = (a[idx] > 0.0f) ? a[idx] : 0.0f;
    }
}

void relu_forward_cuda(const float* a, float* out, int size) {
    int grid = getGridSize(size);
    relu_forward_kernel<<<grid, 256>>>(a, out, size);
    hipDeviceSynchronize();
}

__global__ void relu_backward_kernel(const float* grad_out, const float* a, float* grad_a, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        grad_a[idx] += (a[idx] > 0.0f) ? grad_out[idx] : 0.0f;
    }
}

void relu_backward_cuda(const float* grad_out, const float* a, float* grad_a, int size) {
    int grid = getGridSize(size);
    relu_backward_kernel<<<grid, 256>>>(grad_out, a, grad_a, size);
    hipDeviceSynchronize();
}

// -------------------- Sum --------------------
// forward: sum all elements of a into out[0]
__global__ void sum_forward_kernel(const float* a, float* out, int size) {
    // We'll do a simple parallel reduction. For simplicity, we might do a naive approach.
    // In real code, use a proper reduction for performance.
    __shared__ float sdata[256];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float val = 0.0f;
    if (idx < size) val = a[idx];
    sdata[threadIdx.x] = val;
    __syncthreads();

    // Reduce within block
    for (int s = blockDim.x/2; s>0; s>>=1) {
        if (threadIdx.x < s) {
            sdata[threadIdx.x] += sdata[threadIdx.x + s];
        }
        __syncthreads();
    }

    // block 0 accumulates into out[0]
    if (threadIdx.x == 0) {
        atomicAdd(out, sdata[0]);
    }
}

void sum_forward_cuda(const float* a, float* out, int size) {
    // Initialize out[0] = 0
    hipMemset(out, 0, sizeof(float));
    int grid = getGridSize(size);
    sum_forward_kernel<<<grid, 256>>>(a, out, size);
    hipDeviceSynchronize();
}

// backward: grad_out is shape [1], spread to all elements of grad_a
__global__ void sum_backward_kernel(const float* grad_out, float* grad_a, int size) {
    float g = grad_out[0];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        grad_a[idx] += g;
    }
}

void sum_backward_cuda(const float* grad_out, float* grad_a, int size) {
    int grid = getGridSize(size);
    sum_backward_kernel<<<grid, 256>>>(grad_out, grad_a, size);
    hipDeviceSynchronize();
}

// -------------------- Stack --------------------
// stack forward: inputs is array of pointers, each input shape [1], output shape [num_inputs]
// For simplicity: we assume inputs pointers are device pointers accessible here.
__global__ void stack_forward_kernel(const float** inputs, float* out, int num_inputs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // each input is scalar. idx < num_inputs
    if (idx < num_inputs) {
        out[idx] = inputs[idx][0];
    }
}

void stack_forward_cuda(const float** inputs, float* out, int num_inputs) {
    int grid = getGridSize(num_inputs);
    stack_forward_kernel<<<grid, 256>>>(inputs, out, num_inputs);
    hipDeviceSynchronize();
}

// stack backward: grad_out shape [num_inputs], distribute each grad_out[i] to grad_ins[i][0]
__global__ void stack_backward_kernel(const float* grad_out, float** grad_ins, int num_inputs) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_inputs) {
        grad_ins[idx][0] += grad_out[idx];
    }
}

void stack_backward_cuda(const float* grad_out, float** grad_ins, int num_inputs) {
    int grid = getGridSize(num_inputs);
    stack_backward_kernel<<<grid, 256>>>(grad_out, grad_ins, num_inputs);
    hipDeviceSynchronize();
}
